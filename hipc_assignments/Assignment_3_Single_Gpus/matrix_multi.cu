//The simple version of the Matrix _ multiplication:
//AUTHOR: SAURAV RAI
//REGD NO: 17558
#include<hip/hip_runtime.h>
#include<stdio.h>
#define BLOCKSIZE 32
void MatrixMultiplication(float *,float *,float *,int);

int main(int argc,char const*argv[]) {
        const int Width = atoi(argv[1]);
    float *M,*N,*P;
    int size = Width*Width*sizeof(float);
   
    hipHostMalloc((void **)&M,size, hipHostMallocDefault);
    hipHostMalloc((void **)&N,size, hipHostMallocDefault);
    hipHostMalloc((void **)&P,size, hipHostMallocDefault);
    
     for(int i = 0; i < (Width*Width) ; i++)
        {
          M[i] = 1;
          N[i] = 1;
          P[i] = 0;
        }
  
    MatrixMultiplication(M, N, P, Width);
    for(int i = 0; i < (Width*Width) ; i++)
      {
        printf("%f \n", P[i]);
      }
    
   hipFree(M);
   hipFree(N);
   hipFree(P);
    return 0;
}

//Matrix multiplication kernel - thread specification
__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int Width)
   {

    //2D Thread ID
    int column = blockIdx.x * BLOCKSIZE + threadIdx.x;
    int row = blockIdx.y * BLOCKSIZE +threadIdx.y;

    //Pvalue stores the Pd element that is computed by the thread
    float Pvalue = 0;

   if(row > Width || column > Width )
      return;
    else
     {  

        for (int k = 0; k < Width ; ++k) 
           {
              Pvalue +=  Md[row *Width + k] * Nd[k *Width + column];
           }
        
       Pd[ row*Width + column] = Pvalue;
    }
  }

void MatrixMultiplication(float *M, float *N, float *P, int Width) {
    int size = Width*Width*sizeof(float);
    float *Md, *Nd, *Pd;

    //Transfer M and N to device memory
    hipMalloc((void**)&Md, size);
    hipMemcpy(Md,M,size,hipMemcpyHostToDevice);
    hipMalloc((void**)&Nd, size);
    hipMemcpy(Nd,N,size,hipMemcpyHostToDevice);

    //Allocate P on the device
    hipMalloc((void**)&Pd,size);
    unsigned int grid_rows = (Width + BLOCKSIZE -1 ) / BLOCKSIZE;
    unsigned int grid_cols = (Width + BLOCKSIZE -1 ) / BLOCKSIZE;	
     
 
    //Setup the execution configuration
    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
    dim3 dimGrid( grid_rows , grid_cols);

    //Launch the device computation threads!
    MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Nd,Pd,Width);

    //Transfer P from device to host
    hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);

    //Free device matrices
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);

}
