//THE SHARED MEMORY PROGRAM FOR MATRIX MULTIPLICATION 
//AUTHOR : SAURAV RAI
//REGD NO: 17558
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define blockD 32   

#define TILE_DIM 32   

  __global__ void MatrixMulKernel(float* , float* , float*,int );
 void MatrixMultiplication(float *, float *, float *,int );


 int main(int argc , const char * argv[])
    {

    const int Width = atoi(argv[1]);
    int size = Width *  Width * sizeof(float);
    
    float *M, *N, *P ;   

    // allocate memory on the CPU
    hipHostMalloc((void **)&M , size, hipHostMallocDefault);
    hipHostMalloc((void **)&N , size, hipHostMallocDefault);
    hipHostMalloc((void **)&P , size, hipHostMallocDefault);
 
 

      // initialize the matrices
   for (int y=0; y < Width; y++)
     {
        for (int x=0; x < Width; x++)
         {
            M[y * Width  + x] = 1; //x + y*wA; 
         }
     }

    for (int y=0; y< Width; y++)
       {
        for (int x=0; x< Width; x++)
          {
             N[y * Width + x] = 1; //x + y*wB; 
          }
       }


    MatrixMultiplication(M, N, P, Width);
   
   for(int i = 0; i < Width * Width ; i++)
	{ 
          printf("%f\n",P[i]);
        } 

   // free the memory allocated on the CPU
    hipFree( M );
    hipFree( N );
    hipFree( P ); 

    return 0;
  }





 __global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
   
{
    float CValue = 0;
    int sum =0;
    int Row = blockIdx.y*TILE_DIM + threadIdx.y;
    int Col = blockIdx.x*TILE_DIM + threadIdx.x;

    __shared__ float Mds[TILE_DIM][TILE_DIM];
    __shared__ float Nds[TILE_DIM][TILE_DIM];

    for (int k = 0; k < (TILE_DIM + Width - 1)/TILE_DIM; k++) {

         if (k*TILE_DIM + threadIdx.x < Width  && Row < Width)
            Mds[threadIdx.y][threadIdx.x] = Md[Row * Width + k*TILE_DIM + threadIdx.x];
         else
             Mds[threadIdx.y][threadIdx.x] = 0.0;

         if (k*TILE_DIM + threadIdx.y < Width && Col < Width)
             Nds[threadIdx.y][threadIdx.x] = Nd[(k*TILE_DIM + threadIdx.y) * Width + Col];
         else
             Nds[threadIdx.y][threadIdx.x] = 0.0;

         __syncthreads();

         if(Row > Width || Col > Width)
           return;

         else
           {
            for (int n = 0; n < TILE_DIM; ++n)
             {
                sum += Mds[threadIdx.y][n] * Nds[n][threadIdx.x];
             }
           CValue = sum;
         __syncthreads();
    }

    if (Row < Width && Col < Width)
        Pd[((blockIdx.y * blockDim.y + threadIdx.y)*Width) +  (blockIdx.x * blockDim.x)+ threadIdx.x] = CValue;
    }
 }





void MatrixMultiplication(float *M, float *N, float *P, int Width)
   {
   int size = Width * Width * sizeof(float);
	   // int size_max = 2 * Width * sizeof(float);
    float *Md, *Nd, *Pd ; 

    // allocate memory on the GPU
    hipMalloc((void**)&Md, size);
    hipMalloc((void**)&Nd, size);
    hipMalloc((void**)&Pd, size);

    // transfer M and N to device memory
    hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
    hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);

    unsigned int grid_rows= (Width + blockD-1)/blockD ;
    unsigned int grid_cols= (Width + blockD -1)/blockD;
 
   // kernel invocation code
    dim3 dimBlock(blockD, blockD);
    dim3 dimGrid( grid_rows,grid_cols);

    //Execute Kernel
    MatrixMulKernel<<<dimGrid, dimBlock>>>( Md, Nd, Pd, Width);

    // transfer P from device    
    
    hipMemcpy(P,Pd, size,hipMemcpyDeviceToHost);

    // free the memory allocated on the GPU
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
    }
